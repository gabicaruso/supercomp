#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>

struct sub
{
    __host__ __device__
    double operator()(const double& x, const double& y) {
        return x - y;
    }
};

// struct media
// {
//     int N;
//     double *aapl, *msft;
//     thrust::device_vector<double> aapl_gpu(aapl, aapl + N);
//     thrust::device_vector<double> msft_gpu(msft, msft + N);

//     media(int n, double aapl, double msft): N(n), aapl(n), msft(n){};
//     // media(int n, double aapl, double msft): N(n), aapl_gpu(n), msft_gpu(n){};
//     __host__ __device__
//     double operator()(const double& aapl, const double& msft) {
//         return (aapl - msft) / N;
//     }
// };

struct variancia
{
    double mean;
    int N;

    variancia(double m, int n): mean(m), N(n){};
    __host__ __device__
    double operator()(const double& x) {
        return pow((x - mean), 2) / N;
    }
};

int main() {
    int N = 2518;
    thrust::device_vector<double> AAPL, MSFT, MEAN_DIF(N, 0), VAR_DIF(N, 0);
    double stocks_AAPL, stocks_MSFT;

    for (int i = 0; i < N; i++)
    {
        std::cin >> stocks_AAPL >> stocks_MSFT;
        AAPL.push_back(stocks_AAPL);
        MSFT.push_back(stocks_MSFT);
    }

    // thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    // double sum = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(), 0.0, thrust::plus<double>());
    // double mean = sum/N;

    // double mean = thrust::transform_reduce(AAPL.begin(), AAPL.end(), MSFT.begin(), media(N), 0.0, thrust::plus<double>()); ?
    
    thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), sub());
    double sum = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(), 0.0, thrust::plus<double>());
    double mean = sum/N;
    std::cout << "Média: " << fabs(mean) << "\n";


    // thrust::transform(MEAN_DIF.begin(), MEAN_DIF.end(), thrust::constant_iterator<double>(mean), VAR_DIF.begin(), thrust::minus<double>());
    // thrust::transform(VAR_DIF.begin(), VAR_DIF.end(), VAR_DIF.begin(), VAR_DIF.begin(), thrust::multiplies<double>());
    // double varSum = thrust::reduce(VAR_DIF.begin(), VAR_DIF.end(), 0.0, thrust::plus<double>());
    // double var = varSum/N;

    double var = thrust::transform_reduce(MEAN_DIF.begin(), MEAN_DIF.end(), variancia(mean, N), 0.0, thrust::plus<double>());
    std::cout << "Variância: " << var << "\n";

    // std::cout << "Device vector MEAN_DIF:" << "\n";
    // for (auto i = MEAN_DIF.begin(); i != MEAN_DIF.end(); i++)
    // {
    //     std::cout << *i << "\n";
    // }
    // std::cout << "\n";
}
