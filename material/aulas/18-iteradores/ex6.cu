#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>

struct is_positive
{
  __host__ __device__
  bool operator()(const double &x)
  {
    return x > 0;
  }
};

struct is_negative
{
  __host__ __device__
  bool operator()(const double &x)
  {
    return x < 0;
  }
};

int main() {
    int N = 2518;
    thrust::host_vector<double> stocks_cpu;
    double stks, mean;

    for (int i = 0; i < N; i++)
    {
        std::cin >> stks;
        stocks_cpu.push_back(stks);
    }

    thrust::device_vector<double> stocks_gpu(stocks_cpu), ganho_diario(N-1, 0);

    thrust::transform(stocks_gpu.begin() + 1, stocks_gpu.end(), stocks_gpu.begin(), ganho_diario.begin(), thrust::minus<double>());
    int valUp = thrust::count_if(ganho_diario.begin(), ganho_diario.end(), is_positive());
    thrust::replace_if(ganho_diario.begin(), ganho_diario.end(), is_negative(), 0);

    double sum = thrust::reduce(ganho_diario.begin(), ganho_diario.end(), 0.0, thrust::plus<double>());
    mean = sum/valUp;

    std::cout << "Soma: " << sum << "\n";
    std::cout << "Média do aumento: " << fabs(mean) << "\n";

    // std::cout << "Ganho diário:" << "\n";
    // for (auto i = ganho_diario.begin(); i != ganho_diario.end(); i++)
    // {
    //     std::cout << *i << "\n";
    // }

    // std::cout << valUp << "\n";
}
