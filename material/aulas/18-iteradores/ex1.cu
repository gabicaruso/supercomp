#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>

int main() {
    int N = 2518;
    thrust::device_vector<double> AAPL, MSFT, MEAN_DIF(N, 0), U_VEC(N, 0), VAR_DIF(N, 0);
    double stocks_AAPL, stocks_MSFT, mean, var;

    for (int i = 0; i < N; i++)
    {
        std::cin >> stocks_AAPL >> stocks_MSFT;
        AAPL.push_back(stocks_AAPL);
        MSFT.push_back(stocks_MSFT);
    }

    thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    double sum = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(), 0.0, thrust::plus<double>());
    mean = sum/N;

    std::cout << "Média: " << fabs(mean) << "\n";

    thrust::fill(U_VEC.begin(), U_VEC.end(), mean);
    thrust::transform(MEAN_DIF.begin(), MEAN_DIF.end(), U_VEC.begin(), VAR_DIF.begin(), thrust::minus<double>());
    thrust::transform(VAR_DIF.begin(), VAR_DIF.end(), VAR_DIF.begin(), VAR_DIF.begin(), thrust::multiplies<double>());
    double varSum = thrust::reduce(VAR_DIF.begin(), VAR_DIF.end(), 0.0, thrust::plus<double>());
    var = varSum/N;

    std::cout << "Variância: " << var << "\n";
}
