#include "hip/hip_runtime.h"
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <iostream>

int main() {
    int N = 2518;
    thrust::device_vector<double> AAPL, MSFT, MEAN_DIF(N, 0);
    double stocks_AAPL, stocks_MSFT, mean;

    for (int i = 0; i < N; i++)
    {
        std::cin >> stocks_AAPL >> stocks_MSFT;
        AAPL.push_back(stocks_AAPL);
        MSFT.push_back(stocks_MSFT);
    }

    thrust::transform(AAPL.begin(), AAPL.end(), MSFT.begin(), MEAN_DIF.begin(), thrust::minus<double>());
    double sum = thrust::reduce(MEAN_DIF.begin(), MEAN_DIF.end(), 0.0, thrust::plus<double>());
    mean = sum/N;
    std::cout << "Média: " << fabs(mean) << "\n";

    // std::cout << "Device vector MEAN_DIF:" << "\n";
    // for (auto i = MEAN_DIF.begin(); i != MEAN_DIF.end(); i++)
    // {
    //     std::cout << *i << "\n";
    // }
    // std::cout << "\n";
    
    // std::cout << "Device vector AAPL:" << "\n";
    // for (auto i = AAPL.begin(); i != AAPL.end(); i++)
    // {
    //     std::cout << *i << "\n";
    // }
    // std::cout << "\n";

    // std::cout << "Device vector MSFT:" << "\n";
    // for (auto i = MSFT.begin(); i != MSFT.end(); i++)
    // {
    //     std::cout << *i << "\n";
    // }
    // std::cout << "\n";
}
